#include <hip/hip_runtime.h>

// cuda kernel function
__global__ void addKernel(int *a, int *b, int *c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

// Export the function to be called from TCC
extern "C" __declspec(dllexport) void cuda_add(int *a, int *b, int *c) {
    int size = 10;
    int *d_a, *d_b, *d_c; // device copies of a, b, c

    // Allocate memory on the GPU
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on GPU
    addKernel<<<1, size>>>(d_a, d_b, d_c, size);

    // Copy result back to host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // need to call this to avoid lingering cuda overhead even after freeing the dll itself
    hipDeviceReset(); // reset the gpu and free all related resources -> 8mb => 2mb again -> this frees all the overhead
}
